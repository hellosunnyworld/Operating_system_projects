﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {

  for (int i = 0; i < input_size; i++){
    vm_write(vm, i, input[i]);
    for (int j = 0; j < 1024; j++){
      if (i % 32==0);
        //printf("%x\n", vm->invert_page_table[j]);
    }
      
    //printf("\n\n\n\n\n================================\n");
  }

  for (int i = input_size - 1; i >= input_size - 32769; i--)
    int value = vm_read(vm, i);

  vm_snapshot(vm, results, 0, input_size);
}
